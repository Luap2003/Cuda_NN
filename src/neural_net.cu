// neural_net.cu
#include "../include/neural_net.h"
#include <hip/hip_runtime.h>

NeuralNetwork* create_neural_net(int num_layers) {
    NeuralNetwork *network = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    network->num_layers = num_layers;
    network->layers = (Layer**)malloc(num_layers * sizeof(Layer*));
    return network;
}

void add_layer_to_neural_net(NeuralNetwork *network, Layer *layer, int index) {
    if (index >= 0 && index < network->num_layers) {
        network->layers[index] = layer;
    }
}

void free_neural_net(NeuralNetwork *network) {
    for (int i = 0; i < network->num_layers; ++i) {
        free_layer(network->layers[i]);
    }
    free(network->layers);
    free(network);
}
