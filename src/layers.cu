#include "hip/hip_runtime.h"
// layers.cu
#include "../include/layers.h"
#include "../include/activations.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h> 

Layer* create_dense_layer(int input_size, int output_size, const char *activation) {
    Layer *layer = (Layer*)malloc(sizeof(Layer));

    // Set layer properties
    layer->type = strdup("dense");  // Duplicate string to avoid pointer issues
    layer->activation = strdup(activation);
    layer->input_size = input_size;
    layer->output_size = output_size;

    // Allocate host memory for weights and biases
    layer->weights = (float*)malloc(input_size * output_size * sizeof(float));
    layer->biases = (float*)malloc(output_size * sizeof(float));

    // Xavier Initialization
    float limit = sqrtf(6.0f / (input_size + output_size));
    for (int i = 0; i < input_size * output_size; ++i) {
        layer->weights[i] = ((float)rand() / RAND_MAX) * 2 * limit - limit;
    }
    for (int i = 0; i < output_size; ++i) {
        layer->biases[i] = 0.0f;  // Initialize biases to zero
    }

    // Allocate device memory
    hipMalloc((void**)&(layer->d_weights), input_size * output_size * sizeof(float));
    hipMalloc((void**)&(layer->d_biases), output_size * sizeof(float));

    // Copy weights and biases to device
    hipMemcpy(layer->d_weights, layer->weights, input_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(layer->d_biases, layer->biases, output_size * sizeof(float), hipMemcpyHostToDevice);

    return layer;
}

__global__ void add_bias(float *d_output, float *d_biases, int batch_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * output_size;
    if (idx < total_elements) {
        int bias_idx = idx % output_size;
        d_output[idx] += d_biases[bias_idx];
    }
}

void forward_layer(Layer *layer, float *d_input, float *d_output, int batch_size) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Dimensions
    int m = batch_size;                // Number of rows in input/output
    int n = layer->output_size;        // Number of columns in output
    int k = layer->input_size;         // Number of columns in input

    float alpha = 1.0f;
    float beta = 0.0f;

    // Perform matrix multiplication: d_output = alpha * d_input * d_weights^T + beta * d_output
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_T, // No transpose on d_input, transpose d_weights
                n, m, k,
                &alpha,
                layer->d_weights, n,     // d_weights^T has dimensions [n x k]
                d_input, k,
                &beta,
                d_output, n);            // d_output has dimensions [n x m]

    int threads_per_block = THREADS_PER_BLOCK;
    int total_elements = batch_size * layer->output_size;
    int num_blocks = (total_elements + threads_per_block - 1) / threads_per_block;

    add_bias<<<num_blocks, threads_per_block>>>(d_output, layer->d_biases, batch_size, layer->output_size);
    hipDeviceSynchronize();

    if (strcmp(layer->activation, "sigmoid") == 0) {
        // Call sigmoid activation function
        sigmoid_kernel<<<num_blocks, threads_per_block>>>(d_output, d_output, total_elements);
        hipDeviceSynchronize();
    } else { 
        // Add other activation functions here using else if
        printf("Activation function not implemented\n");
    }

    hipblasDestroy(handle);

}

void free_layer(Layer *layer) {
    // Free device memory
    hipFree(layer->d_weights);
    hipFree(layer->d_biases);

    // Free host memory
    free(layer->weights);
    free(layer->biases);
    free(layer);
}

void print_layer(Layer *layer) {
    printf("Type: %s\n", layer->type);
    printf("Input Size: %d\n", layer->input_size);
    printf("Output Size: %d\n", layer->output_size);
    printf("Activation: %s\n", layer->activation);

    // Print a snippet of weights and biases
    int num_weights_to_print = 5;
    printf("Weights (first %d values):\n", num_weights_to_print);
    for (int i = 0; i < num_weights_to_print && i < layer->input_size * layer->output_size; ++i) {
        printf("%f ", layer->weights[i]);
    }
    printf("\nBiases (first %d values):\n", num_weights_to_print);
    for (int i = 0; i < num_weights_to_print && i < layer->output_size; ++i) {
        printf("%f ", layer->biases[i]);
    }
    printf("\n");
}