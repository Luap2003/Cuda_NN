#include "hip/hip_runtime.h"
// activations.cu
#include "../include/activations.h"
#include <math.h>

__device__ float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__global__ void sigmoid_kernel(float *input, float *output, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        output[idx] = sigmoid(input[idx]);
    }
}

// Implement other activation functions similarly
